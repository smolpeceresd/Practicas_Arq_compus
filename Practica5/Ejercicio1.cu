
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define N 3

/*Funciones*/
__global__ void kernel_suma(float *v1, float *v2, int dim)
{
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    if (id < dim)
    {
        v1[id] = v1[id] + v2[id];
    }
}

int main()
{
    /*Declaracion de espacios de memoria en host*/
    float *A = (float *)malloc(N * sizeof(float));
    float *B = (float *)malloc(N * sizeof(float));
    float *resolver = (float *)malloc(N * sizeof(float));

    /*Declaración de espacios de memoria en device*/
    float *d_A, *d_B, *d_resolver;
    hipMalloc((void **)&d_A, sizeof(float) * N);
    hipMalloc((void **)&d_B, sizeof(float) * N);
    hipMalloc((void **)&d_resolver, sizeof(float) * N);
    printf("Inicializo el vector A  y B del host\n");
    // Initialize host arrays
    memset(A, 0, sizeof(float) * N);
    memset(B, 0, sizeof(float) * N);
    for (int i = 0; i < N; i++)
    {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }
    printf("Array A: ");
    for (int j = 0; j < N; j++)
    {
        printf("%d,", A[j]);
    }
    printf("\nArray B: ");
    for (int k= 0; k < N; k++)
    {
        printf("%d,", B[k]);
    }

    /*Transferimos los datos del host al device*/
    hipMemcpy(d_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * N, hipMemcpyHostToDevice);
    dim3 nthreads(256);
    dim3 nblocks((N / nthreads.x) + (N % nthreads.x ? 1 : 0));
    /*
     *Las variables dim3 son vectores de 3 enteros que se utilizan para especificar dimensiones.
     Componentes x.y.z. si algun componente no se inicializa es 1.  
    */
    /*Función suma*/
    kernel_suma<<<nblocks.x, nthreads.x>>>(A,B, N);
    hipDeviceSynchronize();
    /*Transfiero los datos de la GPU a la CPU*/
    hipMemcpy(resolver, A, sizeof(float) * N, hipMemcpyDeviceToHost);
    
    printf("\nResolver: ");
    for (int z = 0; z < N; z++)
    {
        printf("%d,", resolver[z]);
    }

    free(A);
    free(B);
    free(resolver);
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}